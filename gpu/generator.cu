
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <ctime>
#include <omp.h>
#include <chrono>
#include <fstream>
#include <string>
#include <filesystem>
#include <cmath>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

const int SEED = 42;
const int THREAD_NO = 32;

const int TIME_STEPS = 10000;
const int save_step = TIME_STEPS/10;

const int GRID_SIZE = 4096; // Adjust system size
const int NCOLS = GRID_SIZE;
const int NROWS = GRID_SIZE;
const int TEMP_STEPS = 20;
const float TEMP_LOW = 1.0/1.30;
const float TEMP_HIGH = 1.0/2.7;

#define IDX(i, j) (i * NCOLS + j)

const float BETA = 1/2.25; // Adjust this parameter for funny things! Critical Value is BETA = 0.2;

int calculateChecksum(bool* data) {
    int checksum = 0;
    for (int i = 0; i < NROWS; i++)
        for (int j = 0; j < NCOLS; j++) {
            checksum += data[IDX(i,j)];
        }
    return checksum;
}

void save_data_to_txt(bool* data, const std::string& filename) {
    std::ofstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Error opening file: " << filename << std::endl;
        return;
    }

    for (int i = 0; i < NROWS; ++i) {
        for (int j = 0; j < NCOLS; ++j) {
            file << data[IDX(i,j)] << " ";
        }
        file << "\n";
    }

    file.close();
}

bool* random_spin_field(int N, int M) {
    bool* field = new bool[N * M];

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < M; ++j) {
            field[IDX(i,j)] = ((rand() % 3)%2); // Randomly choose between -1 and 1
        }
    }

    return field;
}

__global__ void setup_curand_kernel(hiprandState* states, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

__global__ void ising_step(bool* field, float beta, bool is_black, hiprandState* states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int N = 2*idx;// + static_cast<int>(is_black);
    //printf("idx: %d, is_black %d \n", N, is_black);

    //if (N < NROWS * NCOLS) {
        int n = N / NCOLS;
        int m = N % NCOLS;// + ((n%2) ^ is_black);
        m = m + ((n%2) ^ is_black); // it has to be an even by even grid

        // Determine if the cell is black or white based on its position

        // Check if the cell should be updated in this kernel call
        int total = ((field[IDX((n + 1) % NROWS, m)] << 1) - 1 )+
                    ((field[IDX((n - 1 + NROWS) % NROWS, m)] << 1) - 1 )+
                    ((field[IDX(n, (m + 1) % NCOLS)] << 1) - 1 )+
                    ((field[IDX(n, (m - 1 + NCOLS) % NCOLS)] << 1) - 1 );

        float dE = 2 * ((field[IDX(n,m)] << 1) - 1 ) * total;
        //printf("sum: %d", total);
        float rand_val = hiprand_uniform(&states[idx]);

        if (dE <= 0 || (expf(-dE * beta) > rand_val)) {
            //field[idx] *= -1;
            field[IDX(n,m)] = !field[IDX(n,m)];
        }
        
    //}
}





int main() {
    float tempDiff = (TEMP_LOW - TEMP_HIGH)/TEMP_STEPS;
    float ourTemp = TEMP_HIGH;
    std::string directory_name = "PLOT_" + std::to_string(GRID_SIZE);
    std::filesystem::create_directory(directory_name);
    for (int t = 0; t < TEMP_STEPS; t++) {
      //std::string filename = directory_name + "/evolution_" + std::to_string(index) + ".txt";
      srand(SEED);
      bool* field = random_spin_field(NROWS, NCOLS);
      bool* deviceField;
      hipMalloc(&deviceField, NROWS * NCOLS * sizeof(bool));
      hipMemcpy(deviceField, field, NROWS * NCOLS * sizeof(bool), hipMemcpyHostToDevice);
  
      hiprandState* states;
      hipMalloc(&states, NROWS * NCOLS * sizeof(hiprandState));
      setup_curand_kernel<<<(NROWS * NCOLS + THREAD_NO - 1) / (2*THREAD_NO), THREAD_NO>>>(states, SEED);
      bool is_black = false;

      auto start_time1 = std::chrono::high_resolution_clock::now();
      for (int k = 0; k < TIME_STEPS; k++) {
          ising_step<<<(NROWS * NCOLS + THREAD_NO - 1) / (2*THREAD_NO), THREAD_NO>>>(deviceField, ourTemp, is_black, states);
          if (is_black) {
            is_black = false;
          }else {
            is_black = true;
          }
  
  //        if (k % save_step == 0) {
  //          cudaMemcpy(field, deviceField, NROWS * NCOLS * sizeof(bool), cudaMemcpyDeviceToHost);
  //          cudaDeviceSynchronize();
  //          std::string filename = directory_name + "/state_" + std::to_string(k) + ".txt";
  //          save_data_to_txt(field, filename);
  //        }
  
      
      }
      hipDeviceSynchronize();
      auto end_time1 = std::chrono::high_resolution_clock::now();
      auto duration1 = std::chrono::duration_cast<std::chrono::milliseconds>(end_time1 - start_time1).count();
      std::cout << "Update time: " << duration1 << "millseconds" << std::endl;
      hipMemcpy(field, deviceField, NROWS * NCOLS * sizeof(bool), hipMemcpyDeviceToHost);

      std::string filename = directory_name + "/Temperature_" + std::to_string(1/ourTemp) + ".txt";
      std::cout<< t << std::endl;
      save_data_to_txt(field, filename);

      hipFree(deviceField);
      hipFree(states);
      hipDeviceSynchronize();
      std::cout<< "GPU Finished" << std::endl;
      ourTemp = ourTemp + tempDiff;
    }
    return 0;
}
